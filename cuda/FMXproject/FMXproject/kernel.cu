#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "initializer.h"

#include <stdio.h>

#define bit(e, i) ((e >> i) & 1)
#define bit2(e, i) ((e >> (i-1)) & 2)

__global__ void calculate(unsigned int* d_p, uint3** d_t, uint3 start, bool* res) {
	char j;
	short* c = getcandidate(d_t, start);

	unsigned int p1 = d_p[c[0]];
	unsigned int p2 = d_p[c[1]];
	unsigned int p3 = d_p[c[2]];
	unsigned int p4 = d_p[c[3]];
	unsigned int p5 = d_p[c[4]];
	unsigned int p6 = d_p[c[5]];
	unsigned int p7 = d_p[c[6]];

	*res = false;
	
	//validate c1
	for (j = 0; j < 128; j++)
	if ((1 /*ae*/ == (1 - bit2(p1, 1)) * bit(p1, 0) * bit(j, 6) + (1 - bit2(p2, 1)) * bit(p2, 0) * bit(j, 5) + (1 - bit2(p3, 1)) * bit(p3, 0) * bit(j, 4) + (1 - bit2(p4, 1)) * bit(p4, 0) * bit(j, 3) + (1 - bit2(p5, 1)) * bit(p5, 0) * bit(j, 2) + (1 - bit2(p6, 1)) * bit(p6, 0) * bit(j, 1) + (1 - bit2(p7, 1)) * bit(p7, 0) * bit(j, 0))
		&& (0 /*af*/ == (1 - bit2(p1, 3)) * bit(p1, 2) * bit(j, 6) + (1 - bit2(p2, 3)) * bit(p2, 2) * bit(j, 5) + (1 - bit2(p3, 3)) * bit(p3, 2) * bit(j, 4) + (1 - bit2(p4, 3)) * bit(p4, 2) * bit(j, 3) + (1 - bit2(p5, 3)) * bit(p5, 2) * bit(j, 2) + (1 - bit2(p6, 3)) * bit(p6, 2) * bit(j, 1) + (1 - bit2(p7, 3)) * bit(p7, 2) * bit(j, 0))
		&& (0 /*ag*/ == (1 - bit2(p1, 5)) * bit(p1, 4) * bit(j, 6) + (1 - bit2(p2, 5)) * bit(p2, 4) * bit(j, 5) + (1 - bit2(p3, 5)) * bit(p3, 4) * bit(j, 4) + (1 - bit2(p4, 5)) * bit(p4, 4) * bit(j, 3) + (1 - bit2(p5, 5)) * bit(p5, 4) * bit(j, 2) + (1 - bit2(p6, 5)) * bit(p6, 4) * bit(j, 1) + (1 - bit2(p7, 5)) * bit(p7, 4) * bit(j, 0))
		&& (0 /*ah*/ == (1 - bit2(p1, 7)) * bit(p1, 6) * bit(j, 6) + (1 - bit2(p2, 7)) * bit(p2, 6) * bit(j, 5) + (1 - bit2(p3, 7)) * bit(p3, 6) * bit(j, 4) + (1 - bit2(p4, 7)) * bit(p4, 6) * bit(j, 3) + (1 - bit2(p5, 7)) * bit(p5, 6) * bit(j, 2) + (1 - bit2(p6, 7)) * bit(p6, 6) * bit(j, 1) + (1 - bit2(p7, 7)) * bit(p7, 6) * bit(j, 0))
		&& (0 /*be*/ == (1 - bit2(p1, 9)) * bit(p1, 8) * bit(j, 6) + (1 - bit2(p2, 9)) * bit(p2, 8) * bit(j, 5) + (1 - bit2(p3, 9)) * bit(p3, 8) * bit(j, 4) + (1 - bit2(p4, 9)) * bit(p4, 8) * bit(j, 3) + (1 - bit2(p5, 9)) * bit(p5, 8) * bit(j, 2) + (1 - bit2(p6, 9)) * bit(p6, 8) * bit(j, 1) + (1 - bit2(p7, 9)) * bit(p7, 8) * bit(j, 0))
		&& (0 /*bf*/ == (1 - bit2(p1, 11)) * bit(p1, 10) * bit(j, 6) + (1 - bit2(p2, 11)) * bit(p2, 10) * bit(j, 5) + (1 - bit2(p3, 11)) * bit(p3, 10) * bit(j, 4) + (1 - bit2(p4, 11)) * bit(p4, 10) * bit(j, 3) + (1 - bit2(p5, 11)) * bit(p5, 10) * bit(j, 2) + (1 - bit2(p6, 11)) * bit(p6, 10) * bit(j, 1) + (1 - bit2(p7, 11)) * bit(p7, 10) * bit(j, 0))
		&& (1 /*bg*/ == (1 - bit2(p1, 13)) * bit(p1, 12) * bit(j, 6) + (1 - bit2(p2, 13)) * bit(p2, 12) * bit(j, 5) + (1 - bit2(p3, 13)) * bit(p3, 12) * bit(j, 4) + (1 - bit2(p4, 13)) * bit(p4, 12) * bit(j, 3) + (1 - bit2(p5, 13)) * bit(p5, 12) * bit(j, 2) + (1 - bit2(p6, 13)) * bit(p6, 12) * bit(j, 1) + (1 - bit2(p7, 13)) * bit(p7, 12) * bit(j, 0))
		&& (0 /*bh*/ == (1 - bit2(p1, 15)) * bit(p1, 14) * bit(j, 6) + (1 - bit2(p2, 15)) * bit(p2, 14) * bit(j, 5) + (1 - bit2(p3, 15)) * bit(p3, 14) * bit(j, 4) + (1 - bit2(p4, 15)) * bit(p4, 14) * bit(j, 3) + (1 - bit2(p5, 15)) * bit(p5, 14) * bit(j, 2) + (1 - bit2(p6, 15)) * bit(p6, 14) * bit(j, 1) + (1 - bit2(p7, 15)) * bit(p7, 14) * bit(j, 0))
		&& (0 /*ce*/ == (1 - bit2(p1, 17)) * bit(p1, 16) * bit(j, 6) + (1 - bit2(p2, 17)) * bit(p2, 16) * bit(j, 5) + (1 - bit2(p3, 17)) * bit(p3, 16) * bit(j, 4) + (1 - bit2(p4, 17)) * bit(p4, 16) * bit(j, 3) + (1 - bit2(p5, 17)) * bit(p5, 16) * bit(j, 2) + (1 - bit2(p6, 17)) * bit(p6, 16) * bit(j, 1) + (1 - bit2(p7, 17)) * bit(p7, 16) * bit(j, 0))
		&& (0 /*cf*/ == (1 - bit2(p1, 19)) * bit(p1, 18) * bit(j, 6) + (1 - bit2(p2, 19)) * bit(p2, 18) * bit(j, 5) + (1 - bit2(p3, 19)) * bit(p3, 18) * bit(j, 4) + (1 - bit2(p4, 19)) * bit(p4, 18) * bit(j, 3) + (1 - bit2(p5, 19)) * bit(p5, 18) * bit(j, 2) + (1 - bit2(p6, 19)) * bit(p6, 18) * bit(j, 1) + (1 - bit2(p7, 19)) * bit(p7, 18) * bit(j, 0))
		&& (0 /*cg*/ == (1 - bit2(p1, 21)) * bit(p1, 20) * bit(j, 6) + (1 - bit2(p2, 21)) * bit(p2, 20) * bit(j, 5) + (1 - bit2(p3, 21)) * bit(p3, 20) * bit(j, 4) + (1 - bit2(p4, 21)) * bit(p4, 20) * bit(j, 3) + (1 - bit2(p5, 21)) * bit(p5, 20) * bit(j, 2) + (1 - bit2(p6, 21)) * bit(p6, 20) * bit(j, 1) + (1 - bit2(p7, 21)) * bit(p7, 20) * bit(j, 0))
		&& (0 /*ch*/ == (1 - bit2(p1, 23)) * bit(p1, 22) * bit(j, 6) + (1 - bit2(p2, 23)) * bit(p2, 22) * bit(j, 5) + (1 - bit2(p3, 23)) * bit(p3, 22) * bit(j, 4) + (1 - bit2(p4, 23)) * bit(p4, 22) * bit(j, 3) + (1 - bit2(p5, 23)) * bit(p5, 22) * bit(j, 2) + (1 - bit2(p6, 23)) * bit(p6, 22) * bit(j, 1) + (1 - bit2(p7, 23)) * bit(p7, 22) * bit(j, 0))
		&& (0 /*de*/ == (1 - bit2(p1, 25)) * bit(p1, 24) * bit(j, 6) + (1 - bit2(p2, 25)) * bit(p2, 24) * bit(j, 5) + (1 - bit2(p3, 25)) * bit(p3, 24) * bit(j, 4) + (1 - bit2(p4, 25)) * bit(p4, 24) * bit(j, 3) + (1 - bit2(p5, 25)) * bit(p5, 24) * bit(j, 2) + (1 - bit2(p6, 25)) * bit(p6, 24) * bit(j, 1) + (1 - bit2(p7, 25)) * bit(p7, 24) * bit(j, 0))
		&& (0 /*df*/ == (1 - bit2(p1, 27)) * bit(p1, 26) * bit(j, 6) + (1 - bit2(p2, 27)) * bit(p2, 26) * bit(j, 5) + (1 - bit2(p3, 27)) * bit(p3, 26) * bit(j, 4) + (1 - bit2(p4, 27)) * bit(p4, 26) * bit(j, 3) + (1 - bit2(p5, 27)) * bit(p5, 26) * bit(j, 2) + (1 - bit2(p6, 27)) * bit(p6, 26) * bit(j, 1) + (1 - bit2(p7, 27)) * bit(p7, 26) * bit(j, 0))
		&& (0 /*dg*/ == (1 - bit2(p1, 29)) * bit(p1, 28) * bit(j, 6) + (1 - bit2(p2, 29)) * bit(p2, 28) * bit(j, 5) + (1 - bit2(p3, 29)) * bit(p3, 28) * bit(j, 4) + (1 - bit2(p4, 29)) * bit(p4, 28) * bit(j, 3) + (1 - bit2(p5, 29)) * bit(p5, 28) * bit(j, 2) + (1 - bit2(p6, 29)) * bit(p6, 28) * bit(j, 1) + (1 - bit2(p7, 29)) * bit(p7, 28) * bit(j, 0))
		&& (0 /*dh*/ == (1 - bit2(p1, 31)) * bit(p1, 30) * bit(j, 6) + (1 - bit2(p2, 31)) * bit(p2, 30) * bit(j, 5) + (1 - bit2(p3, 31)) * bit(p3, 30) * bit(j, 4) + (1 - bit2(p4, 31)) * bit(p4, 30) * bit(j, 3) + (1 - bit2(p5, 31)) * bit(p5, 30) * bit(j, 2) + (1 - bit2(p6, 31)) * bit(p6, 30) * bit(j, 1) + (1 - bit2(p7, 31)) * bit(p7, 30) * bit(j, 0)))
	{
		goto c2;
	}
	return;

c2: // validate c2
	for (j = 0; j < 128; j++)
	if ((0 /*ae*/ == (1 - bit2(p1, 1)) * bit(p1, 0) * bit(j, 6) + (1 - bit2(p2, 1)) * bit(p2, 0) * bit(j, 5) + (1 - bit2(p3, 1)) * bit(p3, 0) * bit(j, 4) + (1 - bit2(p4, 1)) * bit(p4, 0) * bit(j, 3) + (1 - bit2(p5, 1)) * bit(p5, 0) * bit(j, 2) + (1 - bit2(p6, 1)) * bit(p6, 0) * bit(j, 1) + (1 - bit2(p7, 1)) * bit(p7, 0) * bit(j, 0))
		&& (1 /*af*/ == (1 - bit2(p1, 3)) * bit(p1, 2) * bit(j, 6) + (1 - bit2(p2, 3)) * bit(p2, 2) * bit(j, 5) + (1 - bit2(p3, 3)) * bit(p3, 2) * bit(j, 4) + (1 - bit2(p4, 3)) * bit(p4, 2) * bit(j, 3) + (1 - bit2(p5, 3)) * bit(p5, 2) * bit(j, 2) + (1 - bit2(p6, 3)) * bit(p6, 2) * bit(j, 1) + (1 - bit2(p7, 3)) * bit(p7, 2) * bit(j, 0))
		&& (0 /*ag*/ == (1 - bit2(p1, 5)) * bit(p1, 4) * bit(j, 6) + (1 - bit2(p2, 5)) * bit(p2, 4) * bit(j, 5) + (1 - bit2(p3, 5)) * bit(p3, 4) * bit(j, 4) + (1 - bit2(p4, 5)) * bit(p4, 4) * bit(j, 3) + (1 - bit2(p5, 5)) * bit(p5, 4) * bit(j, 2) + (1 - bit2(p6, 5)) * bit(p6, 4) * bit(j, 1) + (1 - bit2(p7, 5)) * bit(p7, 4) * bit(j, 0))
		&& (0 /*ah*/ == (1 - bit2(p1, 7)) * bit(p1, 6) * bit(j, 6) + (1 - bit2(p2, 7)) * bit(p2, 6) * bit(j, 5) + (1 - bit2(p3, 7)) * bit(p3, 6) * bit(j, 4) + (1 - bit2(p4, 7)) * bit(p4, 6) * bit(j, 3) + (1 - bit2(p5, 7)) * bit(p5, 6) * bit(j, 2) + (1 - bit2(p6, 7)) * bit(p6, 6) * bit(j, 1) + (1 - bit2(p7, 7)) * bit(p7, 6) * bit(j, 0))
		&& (0 /*be*/ == (1 - bit2(p1, 9)) * bit(p1, 8) * bit(j, 6) + (1 - bit2(p2, 9)) * bit(p2, 8) * bit(j, 5) + (1 - bit2(p3, 9)) * bit(p3, 8) * bit(j, 4) + (1 - bit2(p4, 9)) * bit(p4, 8) * bit(j, 3) + (1 - bit2(p5, 9)) * bit(p5, 8) * bit(j, 2) + (1 - bit2(p6, 9)) * bit(p6, 8) * bit(j, 1) + (1 - bit2(p7, 9)) * bit(p7, 8) * bit(j, 0))
		&& (0 /*bf*/ == (1 - bit2(p1, 11)) * bit(p1, 10) * bit(j, 6) + (1 - bit2(p2, 11)) * bit(p2, 10) * bit(j, 5) + (1 - bit2(p3, 11)) * bit(p3, 10) * bit(j, 4) + (1 - bit2(p4, 11)) * bit(p4, 10) * bit(j, 3) + (1 - bit2(p5, 11)) * bit(p5, 10) * bit(j, 2) + (1 - bit2(p6, 11)) * bit(p6, 10) * bit(j, 1) + (1 - bit2(p7, 11)) * bit(p7, 10) * bit(j, 0))
		&& (0 /*bg*/ == (1 - bit2(p1, 13)) * bit(p1, 12) * bit(j, 6) + (1 - bit2(p2, 13)) * bit(p2, 12) * bit(j, 5) + (1 - bit2(p3, 13)) * bit(p3, 12) * bit(j, 4) + (1 - bit2(p4, 13)) * bit(p4, 12) * bit(j, 3) + (1 - bit2(p5, 13)) * bit(p5, 12) * bit(j, 2) + (1 - bit2(p6, 13)) * bit(p6, 12) * bit(j, 1) + (1 - bit2(p7, 13)) * bit(p7, 12) * bit(j, 0))
		&& (1 /*bh*/ == (1 - bit2(p1, 15)) * bit(p1, 14) * bit(j, 6) + (1 - bit2(p2, 15)) * bit(p2, 14) * bit(j, 5) + (1 - bit2(p3, 15)) * bit(p3, 14) * bit(j, 4) + (1 - bit2(p4, 15)) * bit(p4, 14) * bit(j, 3) + (1 - bit2(p5, 15)) * bit(p5, 14) * bit(j, 2) + (1 - bit2(p6, 15)) * bit(p6, 14) * bit(j, 1) + (1 - bit2(p7, 15)) * bit(p7, 14) * bit(j, 0))
		&& (0 /*ce*/ == (1 - bit2(p1, 17)) * bit(p1, 16) * bit(j, 6) + (1 - bit2(p2, 17)) * bit(p2, 16) * bit(j, 5) + (1 - bit2(p3, 17)) * bit(p3, 16) * bit(j, 4) + (1 - bit2(p4, 17)) * bit(p4, 16) * bit(j, 3) + (1 - bit2(p5, 17)) * bit(p5, 16) * bit(j, 2) + (1 - bit2(p6, 17)) * bit(p6, 16) * bit(j, 1) + (1 - bit2(p7, 17)) * bit(p7, 16) * bit(j, 0))
		&& (0 /*cf*/ == (1 - bit2(p1, 19)) * bit(p1, 18) * bit(j, 6) + (1 - bit2(p2, 19)) * bit(p2, 18) * bit(j, 5) + (1 - bit2(p3, 19)) * bit(p3, 18) * bit(j, 4) + (1 - bit2(p4, 19)) * bit(p4, 18) * bit(j, 3) + (1 - bit2(p5, 19)) * bit(p5, 18) * bit(j, 2) + (1 - bit2(p6, 19)) * bit(p6, 18) * bit(j, 1) + (1 - bit2(p7, 19)) * bit(p7, 18) * bit(j, 0))
		&& (0 /*cg*/ == (1 - bit2(p1, 21)) * bit(p1, 20) * bit(j, 6) + (1 - bit2(p2, 21)) * bit(p2, 20) * bit(j, 5) + (1 - bit2(p3, 21)) * bit(p3, 20) * bit(j, 4) + (1 - bit2(p4, 21)) * bit(p4, 20) * bit(j, 3) + (1 - bit2(p5, 21)) * bit(p5, 20) * bit(j, 2) + (1 - bit2(p6, 21)) * bit(p6, 20) * bit(j, 1) + (1 - bit2(p7, 21)) * bit(p7, 20) * bit(j, 0))
		&& (0 /*ch*/ == (1 - bit2(p1, 23)) * bit(p1, 22) * bit(j, 6) + (1 - bit2(p2, 23)) * bit(p2, 22) * bit(j, 5) + (1 - bit2(p3, 23)) * bit(p3, 22) * bit(j, 4) + (1 - bit2(p4, 23)) * bit(p4, 22) * bit(j, 3) + (1 - bit2(p5, 23)) * bit(p5, 22) * bit(j, 2) + (1 - bit2(p6, 23)) * bit(p6, 22) * bit(j, 1) + (1 - bit2(p7, 23)) * bit(p7, 22) * bit(j, 0))
		&& (0 /*de*/ == (1 - bit2(p1, 25)) * bit(p1, 24) * bit(j, 6) + (1 - bit2(p2, 25)) * bit(p2, 24) * bit(j, 5) + (1 - bit2(p3, 25)) * bit(p3, 24) * bit(j, 4) + (1 - bit2(p4, 25)) * bit(p4, 24) * bit(j, 3) + (1 - bit2(p5, 25)) * bit(p5, 24) * bit(j, 2) + (1 - bit2(p6, 25)) * bit(p6, 24) * bit(j, 1) + (1 - bit2(p7, 25)) * bit(p7, 24) * bit(j, 0))
		&& (0 /*df*/ == (1 - bit2(p1, 27)) * bit(p1, 26) * bit(j, 6) + (1 - bit2(p2, 27)) * bit(p2, 26) * bit(j, 5) + (1 - bit2(p3, 27)) * bit(p3, 26) * bit(j, 4) + (1 - bit2(p4, 27)) * bit(p4, 26) * bit(j, 3) + (1 - bit2(p5, 27)) * bit(p5, 26) * bit(j, 2) + (1 - bit2(p6, 27)) * bit(p6, 26) * bit(j, 1) + (1 - bit2(p7, 27)) * bit(p7, 26) * bit(j, 0))
		&& (0 /*dg*/ == (1 - bit2(p1, 29)) * bit(p1, 28) * bit(j, 6) + (1 - bit2(p2, 29)) * bit(p2, 28) * bit(j, 5) + (1 - bit2(p3, 29)) * bit(p3, 28) * bit(j, 4) + (1 - bit2(p4, 29)) * bit(p4, 28) * bit(j, 3) + (1 - bit2(p5, 29)) * bit(p5, 28) * bit(j, 2) + (1 - bit2(p6, 29)) * bit(p6, 28) * bit(j, 1) + (1 - bit2(p7, 29)) * bit(p7, 28) * bit(j, 0))
		&& (0 /*dh*/ == (1 - bit2(p1, 31)) * bit(p1, 30) * bit(j, 6) + (1 - bit2(p2, 31)) * bit(p2, 30) * bit(j, 5) + (1 - bit2(p3, 31)) * bit(p3, 30) * bit(j, 4) + (1 - bit2(p4, 31)) * bit(p4, 30) * bit(j, 3) + (1 - bit2(p5, 31)) * bit(p5, 30) * bit(j, 2) + (1 - bit2(p6, 31)) * bit(p6, 30) * bit(j, 1) + (1 - bit2(p7, 31)) * bit(p7, 30) * bit(j, 0)))
	{
		goto c3;
	}
	return;

c3: // validate c3
	for (j = 0; j < 128; j++)
	if ((0 /*ae*/ == (1 - bit2(p1, 1)) * bit(p1, 0) * bit(j, 6) + (1 - bit2(p2, 1)) * bit(p2, 0) * bit(j, 5) + (1 - bit2(p3, 1)) * bit(p3, 0) * bit(j, 4) + (1 - bit2(p4, 1)) * bit(p4, 0) * bit(j, 3) + (1 - bit2(p5, 1)) * bit(p5, 0) * bit(j, 2) + (1 - bit2(p6, 1)) * bit(p6, 0) * bit(j, 1) + (1 - bit2(p7, 1)) * bit(p7, 0) * bit(j, 0))
		&& (0 /*af*/ == (1 - bit2(p1, 3)) * bit(p1, 2) * bit(j, 6) + (1 - bit2(p2, 3)) * bit(p2, 2) * bit(j, 5) + (1 - bit2(p3, 3)) * bit(p3, 2) * bit(j, 4) + (1 - bit2(p4, 3)) * bit(p4, 2) * bit(j, 3) + (1 - bit2(p5, 3)) * bit(p5, 2) * bit(j, 2) + (1 - bit2(p6, 3)) * bit(p6, 2) * bit(j, 1) + (1 - bit2(p7, 3)) * bit(p7, 2) * bit(j, 0))
		&& (0 /*ag*/ == (1 - bit2(p1, 5)) * bit(p1, 4) * bit(j, 6) + (1 - bit2(p2, 5)) * bit(p2, 4) * bit(j, 5) + (1 - bit2(p3, 5)) * bit(p3, 4) * bit(j, 4) + (1 - bit2(p4, 5)) * bit(p4, 4) * bit(j, 3) + (1 - bit2(p5, 5)) * bit(p5, 4) * bit(j, 2) + (1 - bit2(p6, 5)) * bit(p6, 4) * bit(j, 1) + (1 - bit2(p7, 5)) * bit(p7, 4) * bit(j, 0))
		&& (0 /*ah*/ == (1 - bit2(p1, 7)) * bit(p1, 6) * bit(j, 6) + (1 - bit2(p2, 7)) * bit(p2, 6) * bit(j, 5) + (1 - bit2(p3, 7)) * bit(p3, 6) * bit(j, 4) + (1 - bit2(p4, 7)) * bit(p4, 6) * bit(j, 3) + (1 - bit2(p5, 7)) * bit(p5, 6) * bit(j, 2) + (1 - bit2(p6, 7)) * bit(p6, 6) * bit(j, 1) + (1 - bit2(p7, 7)) * bit(p7, 6) * bit(j, 0))
		&& (0 /*be*/ == (1 - bit2(p1, 9)) * bit(p1, 8) * bit(j, 6) + (1 - bit2(p2, 9)) * bit(p2, 8) * bit(j, 5) + (1 - bit2(p3, 9)) * bit(p3, 8) * bit(j, 4) + (1 - bit2(p4, 9)) * bit(p4, 8) * bit(j, 3) + (1 - bit2(p5, 9)) * bit(p5, 8) * bit(j, 2) + (1 - bit2(p6, 9)) * bit(p6, 8) * bit(j, 1) + (1 - bit2(p7, 9)) * bit(p7, 8) * bit(j, 0))
		&& (0 /*bf*/ == (1 - bit2(p1, 11)) * bit(p1, 10) * bit(j, 6) + (1 - bit2(p2, 11)) * bit(p2, 10) * bit(j, 5) + (1 - bit2(p3, 11)) * bit(p3, 10) * bit(j, 4) + (1 - bit2(p4, 11)) * bit(p4, 10) * bit(j, 3) + (1 - bit2(p5, 11)) * bit(p5, 10) * bit(j, 2) + (1 - bit2(p6, 11)) * bit(p6, 10) * bit(j, 1) + (1 - bit2(p7, 11)) * bit(p7, 10) * bit(j, 0))
		&& (0 /*bg*/ == (1 - bit2(p1, 13)) * bit(p1, 12) * bit(j, 6) + (1 - bit2(p2, 13)) * bit(p2, 12) * bit(j, 5) + (1 - bit2(p3, 13)) * bit(p3, 12) * bit(j, 4) + (1 - bit2(p4, 13)) * bit(p4, 12) * bit(j, 3) + (1 - bit2(p5, 13)) * bit(p5, 12) * bit(j, 2) + (1 - bit2(p6, 13)) * bit(p6, 12) * bit(j, 1) + (1 - bit2(p7, 13)) * bit(p7, 12) * bit(j, 0))
		&& (0 /*bh*/ == (1 - bit2(p1, 15)) * bit(p1, 14) * bit(j, 6) + (1 - bit2(p2, 15)) * bit(p2, 14) * bit(j, 5) + (1 - bit2(p3, 15)) * bit(p3, 14) * bit(j, 4) + (1 - bit2(p4, 15)) * bit(p4, 14) * bit(j, 3) + (1 - bit2(p5, 15)) * bit(p5, 14) * bit(j, 2) + (1 - bit2(p6, 15)) * bit(p6, 14) * bit(j, 1) + (1 - bit2(p7, 15)) * bit(p7, 14) * bit(j, 0))
		&& (1 /*ce*/ == (1 - bit2(p1, 17)) * bit(p1, 16) * bit(j, 6) + (1 - bit2(p2, 17)) * bit(p2, 16) * bit(j, 5) + (1 - bit2(p3, 17)) * bit(p3, 16) * bit(j, 4) + (1 - bit2(p4, 17)) * bit(p4, 16) * bit(j, 3) + (1 - bit2(p5, 17)) * bit(p5, 16) * bit(j, 2) + (1 - bit2(p6, 17)) * bit(p6, 16) * bit(j, 1) + (1 - bit2(p7, 17)) * bit(p7, 16) * bit(j, 0))
		&& (0 /*cf*/ == (1 - bit2(p1, 19)) * bit(p1, 18) * bit(j, 6) + (1 - bit2(p2, 19)) * bit(p2, 18) * bit(j, 5) + (1 - bit2(p3, 19)) * bit(p3, 18) * bit(j, 4) + (1 - bit2(p4, 19)) * bit(p4, 18) * bit(j, 3) + (1 - bit2(p5, 19)) * bit(p5, 18) * bit(j, 2) + (1 - bit2(p6, 19)) * bit(p6, 18) * bit(j, 1) + (1 - bit2(p7, 19)) * bit(p7, 18) * bit(j, 0))
		&& (0 /*cg*/ == (1 - bit2(p1, 21)) * bit(p1, 20) * bit(j, 6) + (1 - bit2(p2, 21)) * bit(p2, 20) * bit(j, 5) + (1 - bit2(p3, 21)) * bit(p3, 20) * bit(j, 4) + (1 - bit2(p4, 21)) * bit(p4, 20) * bit(j, 3) + (1 - bit2(p5, 21)) * bit(p5, 20) * bit(j, 2) + (1 - bit2(p6, 21)) * bit(p6, 20) * bit(j, 1) + (1 - bit2(p7, 21)) * bit(p7, 20) * bit(j, 0))
		&& (0 /*ch*/ == (1 - bit2(p1, 23)) * bit(p1, 22) * bit(j, 6) + (1 - bit2(p2, 23)) * bit(p2, 22) * bit(j, 5) + (1 - bit2(p3, 23)) * bit(p3, 22) * bit(j, 4) + (1 - bit2(p4, 23)) * bit(p4, 22) * bit(j, 3) + (1 - bit2(p5, 23)) * bit(p5, 22) * bit(j, 2) + (1 - bit2(p6, 23)) * bit(p6, 22) * bit(j, 1) + (1 - bit2(p7, 23)) * bit(p7, 22) * bit(j, 0))
		&& (0 /*de*/ == (1 - bit2(p1, 25)) * bit(p1, 24) * bit(j, 6) + (1 - bit2(p2, 25)) * bit(p2, 24) * bit(j, 5) + (1 - bit2(p3, 25)) * bit(p3, 24) * bit(j, 4) + (1 - bit2(p4, 25)) * bit(p4, 24) * bit(j, 3) + (1 - bit2(p5, 25)) * bit(p5, 24) * bit(j, 2) + (1 - bit2(p6, 25)) * bit(p6, 24) * bit(j, 1) + (1 - bit2(p7, 25)) * bit(p7, 24) * bit(j, 0))
		&& (0 /*df*/ == (1 - bit2(p1, 27)) * bit(p1, 26) * bit(j, 6) + (1 - bit2(p2, 27)) * bit(p2, 26) * bit(j, 5) + (1 - bit2(p3, 27)) * bit(p3, 26) * bit(j, 4) + (1 - bit2(p4, 27)) * bit(p4, 26) * bit(j, 3) + (1 - bit2(p5, 27)) * bit(p5, 26) * bit(j, 2) + (1 - bit2(p6, 27)) * bit(p6, 26) * bit(j, 1) + (1 - bit2(p7, 27)) * bit(p7, 26) * bit(j, 0))
		&& (1 /*dg*/ == (1 - bit2(p1, 29)) * bit(p1, 28) * bit(j, 6) + (1 - bit2(p2, 29)) * bit(p2, 28) * bit(j, 5) + (1 - bit2(p3, 29)) * bit(p3, 28) * bit(j, 4) + (1 - bit2(p4, 29)) * bit(p4, 28) * bit(j, 3) + (1 - bit2(p5, 29)) * bit(p5, 28) * bit(j, 2) + (1 - bit2(p6, 29)) * bit(p6, 28) * bit(j, 1) + (1 - bit2(p7, 29)) * bit(p7, 28) * bit(j, 0))
		&& (0 /*dh*/ == (1 - bit2(p1, 31)) * bit(p1, 30) * bit(j, 6) + (1 - bit2(p2, 31)) * bit(p2, 30) * bit(j, 5) + (1 - bit2(p3, 31)) * bit(p3, 30) * bit(j, 4) + (1 - bit2(p4, 31)) * bit(p4, 30) * bit(j, 3) + (1 - bit2(p5, 31)) * bit(p5, 30) * bit(j, 2) + (1 - bit2(p6, 31)) * bit(p6, 30) * bit(j, 1) + (1 - bit2(p7, 31)) * bit(p7, 30) * bit(j, 0)))
	{
		goto c4;
	}
	return;

c4: // validate c4
	for (j = 0; j < 128; j++)
	if ((0 /*ae*/ == (1 - bit2(p1, 1)) * bit(p1, 0) * bit(j, 6) + (1 - bit2(p2, 1)) * bit(p2, 0) * bit(j, 5) + (1 - bit2(p3, 1)) * bit(p3, 0) * bit(j, 4) + (1 - bit2(p4, 1)) * bit(p4, 0) * bit(j, 3) + (1 - bit2(p5, 1)) * bit(p5, 0) * bit(j, 2) + (1 - bit2(p6, 1)) * bit(p6, 0) * bit(j, 1) + (1 - bit2(p7, 1)) * bit(p7, 0) * bit(j, 0))
		&& (0 /*af*/ == (1 - bit2(p1, 3)) * bit(p1, 2) * bit(j, 6) + (1 - bit2(p2, 3)) * bit(p2, 2) * bit(j, 5) + (1 - bit2(p3, 3)) * bit(p3, 2) * bit(j, 4) + (1 - bit2(p4, 3)) * bit(p4, 2) * bit(j, 3) + (1 - bit2(p5, 3)) * bit(p5, 2) * bit(j, 2) + (1 - bit2(p6, 3)) * bit(p6, 2) * bit(j, 1) + (1 - bit2(p7, 3)) * bit(p7, 2) * bit(j, 0))
		&& (0 /*ag*/ == (1 - bit2(p1, 5)) * bit(p1, 4) * bit(j, 6) + (1 - bit2(p2, 5)) * bit(p2, 4) * bit(j, 5) + (1 - bit2(p3, 5)) * bit(p3, 4) * bit(j, 4) + (1 - bit2(p4, 5)) * bit(p4, 4) * bit(j, 3) + (1 - bit2(p5, 5)) * bit(p5, 4) * bit(j, 2) + (1 - bit2(p6, 5)) * bit(p6, 4) * bit(j, 1) + (1 - bit2(p7, 5)) * bit(p7, 4) * bit(j, 0))
		&& (0 /*ah*/ == (1 - bit2(p1, 7)) * bit(p1, 6) * bit(j, 6) + (1 - bit2(p2, 7)) * bit(p2, 6) * bit(j, 5) + (1 - bit2(p3, 7)) * bit(p3, 6) * bit(j, 4) + (1 - bit2(p4, 7)) * bit(p4, 6) * bit(j, 3) + (1 - bit2(p5, 7)) * bit(p5, 6) * bit(j, 2) + (1 - bit2(p6, 7)) * bit(p6, 6) * bit(j, 1) + (1 - bit2(p7, 7)) * bit(p7, 6) * bit(j, 0))
		&& (0 /*be*/ == (1 - bit2(p1, 9)) * bit(p1, 8) * bit(j, 6) + (1 - bit2(p2, 9)) * bit(p2, 8) * bit(j, 5) + (1 - bit2(p3, 9)) * bit(p3, 8) * bit(j, 4) + (1 - bit2(p4, 9)) * bit(p4, 8) * bit(j, 3) + (1 - bit2(p5, 9)) * bit(p5, 8) * bit(j, 2) + (1 - bit2(p6, 9)) * bit(p6, 8) * bit(j, 1) + (1 - bit2(p7, 9)) * bit(p7, 8) * bit(j, 0))
		&& (0 /*bf*/ == (1 - bit2(p1, 11)) * bit(p1, 10) * bit(j, 6) + (1 - bit2(p2, 11)) * bit(p2, 10) * bit(j, 5) + (1 - bit2(p3, 11)) * bit(p3, 10) * bit(j, 4) + (1 - bit2(p4, 11)) * bit(p4, 10) * bit(j, 3) + (1 - bit2(p5, 11)) * bit(p5, 10) * bit(j, 2) + (1 - bit2(p6, 11)) * bit(p6, 10) * bit(j, 1) + (1 - bit2(p7, 11)) * bit(p7, 10) * bit(j, 0))
		&& (0 /*bg*/ == (1 - bit2(p1, 13)) * bit(p1, 12) * bit(j, 6) + (1 - bit2(p2, 13)) * bit(p2, 12) * bit(j, 5) + (1 - bit2(p3, 13)) * bit(p3, 12) * bit(j, 4) + (1 - bit2(p4, 13)) * bit(p4, 12) * bit(j, 3) + (1 - bit2(p5, 13)) * bit(p5, 12) * bit(j, 2) + (1 - bit2(p6, 13)) * bit(p6, 12) * bit(j, 1) + (1 - bit2(p7, 13)) * bit(p7, 12) * bit(j, 0))
		&& (0 /*bh*/ == (1 - bit2(p1, 15)) * bit(p1, 14) * bit(j, 6) + (1 - bit2(p2, 15)) * bit(p2, 14) * bit(j, 5) + (1 - bit2(p3, 15)) * bit(p3, 14) * bit(j, 4) + (1 - bit2(p4, 15)) * bit(p4, 14) * bit(j, 3) + (1 - bit2(p5, 15)) * bit(p5, 14) * bit(j, 2) + (1 - bit2(p6, 15)) * bit(p6, 14) * bit(j, 1) + (1 - bit2(p7, 15)) * bit(p7, 14) * bit(j, 0))
		&& (0 /*ce*/ == (1 - bit2(p1, 17)) * bit(p1, 16) * bit(j, 6) + (1 - bit2(p2, 17)) * bit(p2, 16) * bit(j, 5) + (1 - bit2(p3, 17)) * bit(p3, 16) * bit(j, 4) + (1 - bit2(p4, 17)) * bit(p4, 16) * bit(j, 3) + (1 - bit2(p5, 17)) * bit(p5, 16) * bit(j, 2) + (1 - bit2(p6, 17)) * bit(p6, 16) * bit(j, 1) + (1 - bit2(p7, 17)) * bit(p7, 16) * bit(j, 0))
		&& (1 /*cf*/ == (1 - bit2(p1, 19)) * bit(p1, 18) * bit(j, 6) + (1 - bit2(p2, 19)) * bit(p2, 18) * bit(j, 5) + (1 - bit2(p3, 19)) * bit(p3, 18) * bit(j, 4) + (1 - bit2(p4, 19)) * bit(p4, 18) * bit(j, 3) + (1 - bit2(p5, 19)) * bit(p5, 18) * bit(j, 2) + (1 - bit2(p6, 19)) * bit(p6, 18) * bit(j, 1) + (1 - bit2(p7, 19)) * bit(p7, 18) * bit(j, 0))
		&& (0 /*cg*/ == (1 - bit2(p1, 21)) * bit(p1, 20) * bit(j, 6) + (1 - bit2(p2, 21)) * bit(p2, 20) * bit(j, 5) + (1 - bit2(p3, 21)) * bit(p3, 20) * bit(j, 4) + (1 - bit2(p4, 21)) * bit(p4, 20) * bit(j, 3) + (1 - bit2(p5, 21)) * bit(p5, 20) * bit(j, 2) + (1 - bit2(p6, 21)) * bit(p6, 20) * bit(j, 1) + (1 - bit2(p7, 21)) * bit(p7, 20) * bit(j, 0))
		&& (0 /*ch*/ == (1 - bit2(p1, 23)) * bit(p1, 22) * bit(j, 6) + (1 - bit2(p2, 23)) * bit(p2, 22) * bit(j, 5) + (1 - bit2(p3, 23)) * bit(p3, 22) * bit(j, 4) + (1 - bit2(p4, 23)) * bit(p4, 22) * bit(j, 3) + (1 - bit2(p5, 23)) * bit(p5, 22) * bit(j, 2) + (1 - bit2(p6, 23)) * bit(p6, 22) * bit(j, 1) + (1 - bit2(p7, 23)) * bit(p7, 22) * bit(j, 0))
		&& (0 /*de*/ == (1 - bit2(p1, 25)) * bit(p1, 24) * bit(j, 6) + (1 - bit2(p2, 25)) * bit(p2, 24) * bit(j, 5) + (1 - bit2(p3, 25)) * bit(p3, 24) * bit(j, 4) + (1 - bit2(p4, 25)) * bit(p4, 24) * bit(j, 3) + (1 - bit2(p5, 25)) * bit(p5, 24) * bit(j, 2) + (1 - bit2(p6, 25)) * bit(p6, 24) * bit(j, 1) + (1 - bit2(p7, 25)) * bit(p7, 24) * bit(j, 0))
		&& (0 /*df*/ == (1 - bit2(p1, 27)) * bit(p1, 26) * bit(j, 6) + (1 - bit2(p2, 27)) * bit(p2, 26) * bit(j, 5) + (1 - bit2(p3, 27)) * bit(p3, 26) * bit(j, 4) + (1 - bit2(p4, 27)) * bit(p4, 26) * bit(j, 3) + (1 - bit2(p5, 27)) * bit(p5, 26) * bit(j, 2) + (1 - bit2(p6, 27)) * bit(p6, 26) * bit(j, 1) + (1 - bit2(p7, 27)) * bit(p7, 26) * bit(j, 0))
		&& (0 /*dg*/ == (1 - bit2(p1, 29)) * bit(p1, 28) * bit(j, 6) + (1 - bit2(p2, 29)) * bit(p2, 28) * bit(j, 5) + (1 - bit2(p3, 29)) * bit(p3, 28) * bit(j, 4) + (1 - bit2(p4, 29)) * bit(p4, 28) * bit(j, 3) + (1 - bit2(p5, 29)) * bit(p5, 28) * bit(j, 2) + (1 - bit2(p6, 29)) * bit(p6, 28) * bit(j, 1) + (1 - bit2(p7, 29)) * bit(p7, 28) * bit(j, 0))
		&& (1 /*dh*/ == (1 - bit2(p1, 31)) * bit(p1, 30) * bit(j, 6) + (1 - bit2(p2, 31)) * bit(p2, 30) * bit(j, 5) + (1 - bit2(p3, 31)) * bit(p3, 30) * bit(j, 4) + (1 - bit2(p4, 31)) * bit(p4, 30) * bit(j, 3) + (1 - bit2(p5, 31)) * bit(p5, 30) * bit(j, 2) + (1 - bit2(p6, 31)) * bit(p6, 30) * bit(j, 1) + (1 - bit2(p7, 31)) * bit(p7, 30) * bit(j, 0)))
	{
		goto sln;
	}
	return;

	sln: // solution found
	*res = true;
}

int main()
{
	hipError_t cudaStatus;
	unsigned int* dev_p = 0;
	uint3** dev_t = 0;
	bool* dev_r = 0;
	bool r = false;

	cudaStatus = initialize(&dev_p, &dev_t, &dev_r);
	if (cudaStatus != hipSuccess) {
		goto CLEANUP;
	}

	printf("START:\n");
	uint3 n = make_uint3(0, 0, 0);

	info("n = (%d, %d, %d): ", n.x, n.y, n.z);
	calculate <<<1, 1>>>(dev_p, dev_t, n, dev_r);

	cudaStatus = hipMemcpy(&r, dev_r, sizeof(bool), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto CLEANUP;
	}

	printf("%s\n", r ? "TRUE" : "FALSE");

CLEANUP:
	hipFree(dev_p);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}