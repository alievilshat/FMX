#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "initializer.h"
#include "utils.h"

#include <stdio.h>

#define GRID_SIZE 10000
#define BLOCK_SIZE 256
#define NUMBER_OF_CANDIDATES 1

#define msk(p, p_i, j) (j * (1 - ((p >> p_i) & 2)) * ((p >> p_i) & 1))

__host__ __device__ void calculate(short* c, unsigned int* d_p, bool* res) {
	short j0, j1, j2, j3, j4, j5, j6;

	//for (int i = 0; i++ < NUMBER_OF_CANDIDATES; nextcandidate(c)) 
	{

		unsigned int p1 = d_p[c[0]];
		unsigned int p2 = d_p[c[1]];
		unsigned int p3 = d_p[c[2]];
		unsigned int p4 = d_p[c[3]];
		unsigned int p5 = d_p[c[4]];
		unsigned int p6 = d_p[c[5]];
		unsigned int p7 = d_p[c[6]];

		//validate c1
		for (j6 = -1; j6 <= 1; j6++)
		for (j5 = -1; j5 <= 1; j5++)
		for (j4 = -1; j4 <= 1; j4++)
		for (j3 = -1; j3 <= 1; j3++)
		for (j2 = -1; j2 <= 1; j2++)
		for (j1 = -1; j1 <= 1; j1++)
		for (j0 = -1; j0 <= 1; j0++) {
			if ((1 /*ae*/ == msk(p1, 0, j6) + msk(p2, 0, j5) + msk(p3, 0, j4) + msk(p4, 0, j3) + msk(p5, 0, j2) + msk(p6, 0, j1) + msk(p7, 0, j0))
				&& (0 /*af*/ == msk(p1, 2, j6) + msk(p2, 2, j5) + msk(p3, 2, j4) + msk(p4, 2, j3) + msk(p5, 2, j2) + msk(p6, 2, j1) + msk(p7, 2, j0))
				&& (0 /*ag*/ == msk(p1, 4, j6) + msk(p2, 4, j5) + msk(p3, 4, j4) + msk(p4, 4, j3) + msk(p5, 4, j2) + msk(p6, 4, j1) + msk(p7, 4, j0))
				&& (0 /*ah*/ == msk(p1, 6, j6) + msk(p2, 6, j5) + msk(p3, 6, j4) + msk(p4, 6, j3) + msk(p5, 6, j2) + msk(p6, 6, j1) + msk(p7, 6, j0))
				&& (0 /*be*/ == msk(p1, 8, j6) + msk(p2, 8, j5) + msk(p3, 8, j4) + msk(p4, 8, j3) + msk(p5, 8, j2) + msk(p6, 8, j1) + msk(p7, 8, j0))
				&& (0 /*bf*/ == msk(p1, 10, j6) + msk(p2, 10, j5) + msk(p3, 10, j4) + msk(p4, 10, j3) + msk(p5, 10, j2) + msk(p6, 10, j1) + msk(p7, 10, j0))
				&& (1 /*bg*/ == msk(p1, 12, j6) + msk(p2, 12, j5) + msk(p3, 12, j4) + msk(p4, 12, j3) + msk(p5, 12, j2) + msk(p6, 12, j1) + msk(p7, 12, j0))
				&& (0 /*bh*/ == msk(p1, 14, j6) + msk(p2, 14, j5) + msk(p3, 14, j4) + msk(p4, 14, j3) + msk(p5, 14, j2) + msk(p6, 14, j1) + msk(p7, 14, j0))
				&& (0 /*ce*/ == msk(p1, 16, j6) + msk(p2, 16, j5) + msk(p3, 16, j4) + msk(p4, 16, j3) + msk(p5, 16, j2) + msk(p6, 16, j1) + msk(p7, 16, j0))
				&& (0 /*cf*/ == msk(p1, 18, j6) + msk(p2, 18, j5) + msk(p3, 18, j4) + msk(p4, 18, j3) + msk(p5, 18, j2) + msk(p6, 18, j1) + msk(p7, 18, j0))
				&& (0 /*cg*/ == msk(p1, 20, j6) + msk(p2, 20, j5) + msk(p3, 20, j4) + msk(p4, 20, j3) + msk(p5, 20, j2) + msk(p6, 20, j1) + msk(p7, 20, j0))
				&& (0 /*ch*/ == msk(p1, 22, j6) + msk(p2, 22, j5) + msk(p3, 22, j4) + msk(p4, 22, j3) + msk(p5, 22, j2) + msk(p6, 22, j1) + msk(p7, 22, j0))
				&& (0 /*de*/ == msk(p1, 24, j6) + msk(p2, 24, j5) + msk(p3, 24, j4) + msk(p4, 24, j3) + msk(p5, 24, j2) + msk(p6, 24, j1) + msk(p7, 24, j0))
				&& (0 /*df*/ == msk(p1, 26, j6) + msk(p2, 26, j5) + msk(p3, 26, j4) + msk(p4, 26, j3) + msk(p5, 26, j2) + msk(p6, 26, j1) + msk(p7, 26, j0))
				&& (0 /*dg*/ == msk(p1, 28, j6) + msk(p2, 28, j5) + msk(p3, 28, j4) + msk(p4, 28, j3) + msk(p5, 28, j2) + msk(p6, 28, j1) + msk(p7, 28, j0))
				&& (0 /*dh*/ == msk(p1, 30, j6) + msk(p2, 30, j5) + msk(p3, 30, j4) + msk(p4, 30, j3) + msk(p5, 30, j2) + msk(p6, 30, j1) + msk(p7, 30, j0)))
			{
				goto c2;
			}
		}
		return;// continue;

	c2: // validate c2
		for (j6 = -1; j6 <= 1; j6++)
		for (j5 = -1; j5 <= 1; j5++)
		for (j4 = -1; j4 <= 1; j4++)
		for (j3 = -1; j3 <= 1; j3++)
		for (j2 = -1; j2 <= 1; j2++)
		for (j1 = -1; j1 <= 1; j1++)
		for (j0 = -1; j0 <= 1; j0++) {
			if ((0 /*ae*/ == msk(p1, 0, j6) + msk(p2, 0, j5) + msk(p3, 0, j4) + msk(p4, 0, j3) + msk(p5, 0, j2) + msk(p6, 0, j1) + msk(p7, 0, j0))
				&& (1 /*af*/ == msk(p1, 2, j6) + msk(p2, 2, j5) + msk(p3, 2, j4) + msk(p4, 2, j3) + msk(p5, 2, j2) + msk(p6, 2, j1) + msk(p7, 2, j0))
				&& (0 /*ag*/ == msk(p1, 4, j6) + msk(p2, 4, j5) + msk(p3, 4, j4) + msk(p4, 4, j3) + msk(p5, 4, j2) + msk(p6, 4, j1) + msk(p7, 4, j0))
				&& (0 /*ah*/ == msk(p1, 6, j6) + msk(p2, 6, j5) + msk(p3, 6, j4) + msk(p4, 6, j3) + msk(p5, 6, j2) + msk(p6, 6, j1) + msk(p7, 6, j0))
				&& (0 /*be*/ == msk(p1, 8, j6) + msk(p2, 8, j5) + msk(p3, 8, j4) + msk(p4, 8, j3) + msk(p5, 8, j2) + msk(p6, 8, j1) + msk(p7, 8, j0))
				&& (0 /*bf*/ == msk(p1, 10, j6) + msk(p2, 10, j5) + msk(p3, 10, j4) + msk(p4, 10, j3) + msk(p5, 10, j2) + msk(p6, 10, j1) + msk(p7, 10, j0))
				&& (0 /*bg*/ == msk(p1, 12, j6) + msk(p2, 12, j5) + msk(p3, 12, j4) + msk(p4, 12, j3) + msk(p5, 12, j2) + msk(p6, 12, j1) + msk(p7, 12, j0))
				&& (1 /*bh*/ == msk(p1, 14, j6) + msk(p2, 14, j5) + msk(p3, 14, j4) + msk(p4, 14, j3) + msk(p5, 14, j2) + msk(p6, 14, j1) + msk(p7, 14, j0))
				&& (0 /*ce*/ == msk(p1, 16, j6) + msk(p2, 16, j5) + msk(p3, 16, j4) + msk(p4, 16, j3) + msk(p5, 16, j2) + msk(p6, 16, j1) + msk(p7, 16, j0))
				&& (0 /*cf*/ == msk(p1, 18, j6) + msk(p2, 18, j5) + msk(p3, 18, j4) + msk(p4, 18, j3) + msk(p5, 18, j2) + msk(p6, 18, j1) + msk(p7, 18, j0))
				&& (0 /*cg*/ == msk(p1, 20, j6) + msk(p2, 20, j5) + msk(p3, 20, j4) + msk(p4, 20, j3) + msk(p5, 20, j2) + msk(p6, 20, j1) + msk(p7, 20, j0))
				&& (0 /*ch*/ == msk(p1, 22, j6) + msk(p2, 22, j5) + msk(p3, 22, j4) + msk(p4, 22, j3) + msk(p5, 22, j2) + msk(p6, 22, j1) + msk(p7, 22, j0))
				&& (0 /*de*/ == msk(p1, 24, j6) + msk(p2, 24, j5) + msk(p3, 24, j4) + msk(p4, 24, j3) + msk(p5, 24, j2) + msk(p6, 24, j1) + msk(p7, 24, j0))
				&& (0 /*df*/ == msk(p1, 26, j6) + msk(p2, 26, j5) + msk(p3, 26, j4) + msk(p4, 26, j3) + msk(p5, 26, j2) + msk(p6, 26, j1) + msk(p7, 26, j0))
				&& (0 /*dg*/ == msk(p1, 28, j6) + msk(p2, 28, j5) + msk(p3, 28, j4) + msk(p4, 28, j3) + msk(p5, 28, j2) + msk(p6, 28, j1) + msk(p7, 28, j0))
				&& (0 /*dh*/ == msk(p1, 30, j6) + msk(p2, 30, j5) + msk(p3, 30, j4) + msk(p4, 30, j3) + msk(p5, 30, j2) + msk(p6, 30, j1) + msk(p7, 30, j0)))
			{
				goto c3;
			}
		}
		return;// continue;

	c3: // validate c3
		for (j6 = -1; j6 <= 1; j6++)
		for (j5 = -1; j5 <= 1; j5++)
		for (j4 = -1; j4 <= 1; j4++)
		for (j3 = -1; j3 <= 1; j3++)
		for (j2 = -1; j2 <= 1; j2++)
		for (j1 = -1; j1 <= 1; j1++)
		for (j0 = -1; j0 <= 1; j0++) {
			if ((0 /*ae*/ == msk(p1, 0, j6) + msk(p2, 0, j5) + msk(p3, 0, j4) + msk(p4, 0, j3) + msk(p5, 0, j2) + msk(p6, 0, j1) + msk(p7, 0, j0))
				&& (0 /*af*/ == msk(p1, 2, j6) + msk(p2, 2, j5) + msk(p3, 2, j4) + msk(p4, 2, j3) + msk(p5, 2, j2) + msk(p6, 2, j1) + msk(p7, 2, j0))
				&& (0 /*ag*/ == msk(p1, 4, j6) + msk(p2, 4, j5) + msk(p3, 4, j4) + msk(p4, 4, j3) + msk(p5, 4, j2) + msk(p6, 4, j1) + msk(p7, 4, j0))
				&& (0 /*ah*/ == msk(p1, 6, j6) + msk(p2, 6, j5) + msk(p3, 6, j4) + msk(p4, 6, j3) + msk(p5, 6, j2) + msk(p6, 6, j1) + msk(p7, 6, j0))
				&& (0 /*be*/ == msk(p1, 8, j6) + msk(p2, 8, j5) + msk(p3, 8, j4) + msk(p4, 8, j3) + msk(p5, 8, j2) + msk(p6, 8, j1) + msk(p7, 8, j0))
				&& (0 /*bf*/ == msk(p1, 10, j6) + msk(p2, 10, j5) + msk(p3, 10, j4) + msk(p4, 10, j3) + msk(p5, 10, j2) + msk(p6, 10, j1) + msk(p7, 10, j0))
				&& (0 /*bg*/ == msk(p1, 12, j6) + msk(p2, 12, j5) + msk(p3, 12, j4) + msk(p4, 12, j3) + msk(p5, 12, j2) + msk(p6, 12, j1) + msk(p7, 12, j0))
				&& (0 /*bh*/ == msk(p1, 14, j6) + msk(p2, 14, j5) + msk(p3, 14, j4) + msk(p4, 14, j3) + msk(p5, 14, j2) + msk(p6, 14, j1) + msk(p7, 14, j0))
				&& (1 /*ce*/ == msk(p1, 16, j6) + msk(p2, 16, j5) + msk(p3, 16, j4) + msk(p4, 16, j3) + msk(p5, 16, j2) + msk(p6, 16, j1) + msk(p7, 16, j0))
				&& (0 /*cf*/ == msk(p1, 18, j6) + msk(p2, 18, j5) + msk(p3, 18, j4) + msk(p4, 18, j3) + msk(p5, 18, j2) + msk(p6, 18, j1) + msk(p7, 18, j0))
				&& (0 /*cg*/ == msk(p1, 20, j6) + msk(p2, 20, j5) + msk(p3, 20, j4) + msk(p4, 20, j3) + msk(p5, 20, j2) + msk(p6, 20, j1) + msk(p7, 20, j0))
				&& (0 /*ch*/ == msk(p1, 22, j6) + msk(p2, 22, j5) + msk(p3, 22, j4) + msk(p4, 22, j3) + msk(p5, 22, j2) + msk(p6, 22, j1) + msk(p7, 22, j0))
				&& (0 /*de*/ == msk(p1, 24, j6) + msk(p2, 24, j5) + msk(p3, 24, j4) + msk(p4, 24, j3) + msk(p5, 24, j2) + msk(p6, 24, j1) + msk(p7, 24, j0))
				&& (0 /*df*/ == msk(p1, 26, j6) + msk(p2, 26, j5) + msk(p3, 26, j4) + msk(p4, 26, j3) + msk(p5, 26, j2) + msk(p6, 26, j1) + msk(p7, 26, j0))
				&& (1 /*dg*/ == msk(p1, 28, j6) + msk(p2, 28, j5) + msk(p3, 28, j4) + msk(p4, 28, j3) + msk(p5, 28, j2) + msk(p6, 28, j1) + msk(p7, 28, j0))
				&& (0 /*dh*/ == msk(p1, 30, j6) + msk(p2, 30, j5) + msk(p3, 30, j4) + msk(p4, 30, j3) + msk(p5, 30, j2) + msk(p6, 30, j1) + msk(p7, 30, j0)))
			{
				goto c4;
			}
		}
		return;// continue;

	c4: // validate c4
		for (j6 = -1; j6 <= 1; j6++)
		for (j5 = -1; j5 <= 1; j5++)
		for (j4 = -1; j4 <= 1; j4++)
		for (j3 = -1; j3 <= 1; j3++)
		for (j2 = -1; j2 <= 1; j2++)
		for (j1 = -1; j1 <= 1; j1++)
		for (j0 = -1; j0 <= 1; j0++) {
			if ((0 /*ae*/ == msk(p1, 0, j6) + msk(p2, 0, j5) + msk(p3, 0, j4) + msk(p4, 0, j3) + msk(p5, 0, j2) + msk(p6, 0, j1) + msk(p7, 0, j0))
				&& (0 /*af*/ == msk(p1, 2, j6) + msk(p2, 2, j5) + msk(p3, 2, j4) + msk(p4, 2, j3) + msk(p5, 2, j2) + msk(p6, 2, j1) + msk(p7, 2, j0))
				&& (0 /*ag*/ == msk(p1, 4, j6) + msk(p2, 4, j5) + msk(p3, 4, j4) + msk(p4, 4, j3) + msk(p5, 4, j2) + msk(p6, 4, j1) + msk(p7, 4, j0))
				&& (0 /*ah*/ == msk(p1, 6, j6) + msk(p2, 6, j5) + msk(p3, 6, j4) + msk(p4, 6, j3) + msk(p5, 6, j2) + msk(p6, 6, j1) + msk(p7, 6, j0))
				&& (0 /*be*/ == msk(p1, 8, j6) + msk(p2, 8, j5) + msk(p3, 8, j4) + msk(p4, 8, j3) + msk(p5, 8, j2) + msk(p6, 8, j1) + msk(p7, 8, j0))
				&& (0 /*bf*/ == msk(p1, 10, j6) + msk(p2, 10, j5) + msk(p3, 10, j4) + msk(p4, 10, j3) + msk(p5, 10, j2) + msk(p6, 10, j1) + msk(p7, 10, j0))
				&& (0 /*bg*/ == msk(p1, 12, j6) + msk(p2, 12, j5) + msk(p3, 12, j4) + msk(p4, 12, j3) + msk(p5, 12, j2) + msk(p6, 12, j1) + msk(p7, 12, j0))
				&& (0 /*bh*/ == msk(p1, 14, j6) + msk(p2, 14, j5) + msk(p3, 14, j4) + msk(p4, 14, j3) + msk(p5, 14, j2) + msk(p6, 14, j1) + msk(p7, 14, j0))
				&& (0 /*ce*/ == msk(p1, 16, j6) + msk(p2, 16, j5) + msk(p3, 16, j4) + msk(p4, 16, j3) + msk(p5, 16, j2) + msk(p6, 16, j1) + msk(p7, 16, j0))
				&& (1 /*cf*/ == msk(p1, 18, j6) + msk(p2, 18, j5) + msk(p3, 18, j4) + msk(p4, 18, j3) + msk(p5, 18, j2) + msk(p6, 18, j1) + msk(p7, 18, j0))
				&& (0 /*cg*/ == msk(p1, 20, j6) + msk(p2, 20, j5) + msk(p3, 20, j4) + msk(p4, 20, j3) + msk(p5, 20, j2) + msk(p6, 20, j1) + msk(p7, 20, j0))
				&& (0 /*ch*/ == msk(p1, 22, j6) + msk(p2, 22, j5) + msk(p3, 22, j4) + msk(p4, 22, j3) + msk(p5, 22, j2) + msk(p6, 22, j1) + msk(p7, 22, j0))
				&& (0 /*de*/ == msk(p1, 24, j6) + msk(p2, 24, j5) + msk(p3, 24, j4) + msk(p4, 24, j3) + msk(p5, 24, j2) + msk(p6, 24, j1) + msk(p7, 24, j0))
				&& (0 /*df*/ == msk(p1, 26, j6) + msk(p2, 26, j5) + msk(p3, 26, j4) + msk(p4, 26, j3) + msk(p5, 26, j2) + msk(p6, 26, j1) + msk(p7, 26, j0))
				&& (0 /*dg*/ == msk(p1, 28, j6) + msk(p2, 28, j5) + msk(p3, 28, j4) + msk(p4, 28, j3) + msk(p5, 28, j2) + msk(p6, 28, j1) + msk(p7, 28, j0))
				&& (1 /*dh*/ == msk(p1, 30, j6) + msk(p2, 30, j5) + msk(p3, 30, j4) + msk(p4, 30, j3) + msk(p5, 30, j2) + msk(p6, 30, j1) + msk(p7, 30, j0)))
			{
				*res = true;
				return;
			}
		}
	}
}

__global__ void g_calculate(unsigned int* d_p, uint3* d_t, uint3 start, bool* res) {

	short c[7];
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	getcandidate(d_t, uint3_add(start, make_uint3(id, 0, 0)), c);
	calculate(c, d_p, res);
}

__global__ void g_getcandidateindex(uint3* d_t, short* res, uint3* n)
{
	getcandidateindex(d_t, res, n);
}

__global__ void g_getcandidate(uint3* d_t, uint3 n, short* res)
{
	getcandidate(d_t, n, res);
}

int main()
{
	hipError_t cudaStatus;
	unsigned int* dev_p = 0;
	uint3* dev_t = 0;
	bool* dev_r = 0;
	bool r = false;

	cudaStatus = initialize(&dev_p, &dev_t, &dev_r);
	if (cudaStatus != hipSuccess) {
		goto CLEANUP;
	}

	info("START:\n");

	uint3 max = make_uint3(0x1a451e22, 0x4823143b, 0x25);
	uint3 inc = make_uint3(GRID_SIZE * BLOCK_SIZE * NUMBER_OF_CANDIDATES, 0, 0);
	uint3 start = make_uint3(0, 0, 0);

	uint3 strassen = make_uint3(3261699961, 1784383582, 4); // { 57, 160, 350, 1050, 1311, 1771, 2961 }

	for (uint3 n = start; uint3_cmp(n, max) < 0; n = add(n, inc)) {

		g_calculate << <GRID_SIZE, BLOCK_SIZE>> >(dev_p, dev_t, n, dev_r);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			info("(%u, %u, %u): cuda execution failed!\n", n.x, n.y, n.z);
			goto CLEANUP;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			info("(%u, %u, %u): hipDeviceSynchronize returned error code %d after launching addKernel!\n", n.x, n.y, n.z, cudaStatus);
			goto CLEANUP;
		}

		cudaStatus = hipMemcpy(&r, dev_r, sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			info("(%u, %u, %u): hipMemcpy failed!\n", n.x, n.y, n.z);
			goto CLEANUP;
		}

		info("(%u, %u, %u): %s\n", n.x, n.y, n.z, r ? "TRUE" : "FALSE");
		if (r) break;
	}

CLEANUP:
	hipFree(dev_p);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}