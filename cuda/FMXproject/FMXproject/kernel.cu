#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "initializer.h"

#include <stdio.h>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	hipError_t cudaStatus;
	unsigned int* dev_p = 0;
	unsigned int* dev_t = 0;

	cudaStatus = initialize(&dev_p, &dev_t);
	if (cudaStatus != hipSuccess) {
		goto CLEANUP;
	}

	printf("DONE\n");

CLEANUP:
	hipFree(dev_p);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

/*

// Launch a kernel on the GPU with one thread for each element.
addKernel <<<1, size>>>(dev_c, dev_a, dev_b);
*/
